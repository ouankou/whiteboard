#include "hip/hip_runtime.h"
#include "common.h"
#include "cudaLib.cuh"
#include <omp.h>
#define REAL double

namespace stencil2d_9pt {
	static const int WARP_SIZE = 32;
	static const int FILTER_WIDTH = 5;
	static const int FILTER_HEIGHT = 5;

	void j2d9pt(const REAL* __restrict__ src, REAL* dst, int width, int height, REAL fc, REAL fn0, REAL fs0, REAL fw0, REAL fe0, REAL fn1, REAL fs1, REAL fw1, REAL fe1)
	{
        // 2 gird, 4 block, multiple mini warps
        // mini warp: 8 threads: 4 for data points, 4 as helper
		const int PROCESS_DATA_COUNT = 4;
		const int DATA_CACHE_SIZE = PROCESS_DATA_COUNT + FILTER_HEIGHT - 1;
        int i, j, k;
#pragma omp parallel for num_threads(2)
        for (i = 0; i < height; i++)
        {
#pragma omp parallel for num_threads(4)
            for (j = 0; j < width; j += 4)
            {
                //REAL sum[PROCESS_DATA_COUNT];
                REAL sum[DATA_CACHE_SIZE];
#pragma omp parallel for num_threads(8) ordered
                for (k = 0; k < DATA_CACHE_SIZE; k++)
                {
                    int p, idx_x, idx_y;
                    REAL data[DATA_CACHE_SIZE];
                    int tidy = i;
                    int tidx = j + k;
                    int index = tidx * width + tidy;
                    if (tidx < 0)            index -= tidx;
                    else if (tidx >= width)  index -= tidx - width + 1;
                    if (tidy < 0)            index -= tidy*width;
                    else if (tidy >= height) index -= (tidy - height + 1)*width;
                    for (p = 0; p < FILTER_HEIGHT; p++)
                    {
                        /*
                        idx_y = i - 2 + p;
                        if (idx_y < 0)
                        {
                           idx_y = 0;
                        }
                        else if (idx_y >= height)
                        {
                            idx_y = height - 1;
                        };
                        idx_x = j + k;
                        if (idx_x < 0)
                        {
                           idx_x = 0;
                        }
                        else if (idx_x >= width)
                        {
                            idx_x = width - 1;
                        };
                        data[p] = src[idx_y * width + idx_x];
                        */
                        int _tidx = tidx + p;
				        data[p] = src[index];
				        if (_tidx >= 0 && _tidx < width - 1) {
					        index += 1;
				        };
                    };
                    int tid = omp_get_thread_num();
                    sum[tid] = data[2] * fe1;
                    // shuffle
#pragma omp ordered threads
                    if (tid < 8)
                    {
                        sum[tid] = sum[tid + 1];
                    };
                    sum[tid] += data[2] * fe0;
#pragma omp ordered threads
                    if (tid < 8)
                    {
                        sum[tid] = sum[tid + 1];
                    };
                    sum[tid] += data[0] * fn1;
                    sum[tid] += data[1] * fn0;
                    sum[tid] += data[2] * fc;
                    sum[tid] += data[3] * fs0;
                    sum[tid] += data[4] * fs1;
#pragma omp ordered threads
                    if (tid < 8)
                    {
                        sum[tid] = sum[tid + 1];
                    };
                    sum[tid] += data[2] * fw0;
#pragma omp ordered threads
                    if (tid < 8)
                    {
                        sum[tid] = sum[tid + 1];
                    };
                    sum[tid] += data[2] * fw1;
                };
                for (k = 0; k < 4; k++)
                {
                    dst[i * width + j + k] = sum[k];
                };
            };
        };
	}

	template<class DataType, int PROCESS_DATA_COUNT, int BLOCK_SIZE>
	static void Test(int width, int height) {
		const int WARP_COUNT = BLOCK_SIZE >> 5;
		const int WARP_PROCESS_DATA_COUNT = WARP_SIZE - FILTER_WIDTH + 1;
		const int BLOCK_PROCESS_DATA_COUNT = WARP_PROCESS_DATA_COUNT*WARP_COUNT;

		const int nRepeatCount = 1;
		float inc = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		//StopWatchWin watch;
		DataT<DataType> img;
		char szPath[1024] = "";
		sprintf(szPath, "../Lena_%dx%d.raw", width, height);
		bool bRtn = img.Load_uchar(szPath, width, height);
		//sprintf(szPath, "../data/Lena%dx%d.txt", width, height);
		//img.SaveText(szPath);
		if (!bRtn) {
			printf("Load failed : %s, generate random data\n", szPath);
			img.MallocBuffer(width, height);
			for (int i = 0; i < img.width*img.height; i++) {
				img.data[i] = std::rand() % 256;
				//img.data[i] = i/img.width;
			}
		}
		else {
			printf("Load success : %s\n", szPath);
		}
		//sprintf(szPath, "../Lena_%dx%d.raw", width, height);
		//img.SaveRaw(szPath);

		DevBuffer<DataType> devSrc(width, height), devDst(width, height);
		devSrc.CopyFromHost(img.data, img.width, img.width, img.height);
		DataT<DataType> imgDst;
		imgDst.MallocBuffer(width, height);

		//dim3 block_size(BLOCK_SIZE, 1);
		dim3 grid_size(UpDivide(width, BLOCK_PROCESS_DATA_COUNT), UpDivide(height, PROCESS_DATA_COUNT));

		DataType filter[FILTER_HEIGHT][FILTER_WIDTH] = {
			{ 0,  0, 1, 0, 0, },
			{ 0,  0, 2, 0, 0, },
			{ 3,  4, 5, 6, 7, },
			{ 0,  0, 8, 0, 0, },
			{ 0,  0, 9, 0, 0, },
		};

		DataType fc = filter[2][2];
		DataType fn0 = filter[1][2];
		DataType fs0 = filter[3][2];
		DataType fw0 = filter[2][1];
		DataType fe0 = filter[2][3];
		DataType fn1 = filter[0][2];
		DataType fs1 = filter[4][2];
		DataType fw1 = filter[2][0];
		DataType fe1 = filter[2][4];

        //printf("src: %f\n", ((double*)(devSrc.GetData()))[0]);
		hipEventRecord(start, 0);
		for (int s = 0; s < nRepeatCount; s++) {
			//j2d9pt(devSrc.GetData(), devDst.GetData(), width, height, fc, fn0, fs0, fw0, fe0, fn1, fs1, fw1, fe1);
			j2d9pt(img.data, imgDst.data, width, height, fc, fn0, fs0, fw0, fe0, fn1, fs1, fw1, fe1);
		}
		hipDeviceSynchronize();
		//watch.stop();
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		CUDA_CHECK_ERROR;

		//devDst.CopyToHost(imgDst.data, imgDst.width, imgDst.width, imgDst.height);

		hipEventElapsedTime(&inc, start, stop);
		//inc = watch.getAverageTime();
		inc /= (float)nRepeatCount;
		printf("%dx%d , %dx%d , proc_count=%d, cache=%d, BLOCK_SIZE=%d, %f ms , %f fps\n", width, height, FILTER_WIDTH, FILTER_HEIGHT, PROCESS_DATA_COUNT, PROCESS_DATA_COUNT + FILTER_HEIGHT - 1, BLOCK_SIZE, inc, 1000.0 / inc);
		sprintf(szPath, "../Lena_omp_proc_%dx%d.raw", width, height);
		imgDst.SaveRaw(szPath);

		sprintf(szPath, "../Lena_omp_proc_%dx%d.txt", width, height);
		imgDst.SaveText(szPath);

		DataT<DataType> imgVerify;
		imgVerify.MallocBuffer(width, height);
		Convolution(img.data, imgVerify.data, width, height, filter[0], FILTER_WIDTH, FILTER_HEIGHT);
		sprintf(szPath, "../data/Lena_proc_verify_%dx%d.txt", width, height);
		//imgVerify.SaveText(szPath);

		double dif = 0;
		for (int i = 0; i < img.width*img.height; i++) {
			int x = i % img.width;
			int y = i / img.width;
			if (x > FILTER_WIDTH/2 && x < width - FILTER_WIDTH/2 && y > FILTER_HEIGHT/2 && y < height - FILTER_HEIGHT/2)
				dif += abs(imgVerify.data[i] - imgDst.data[i]);
		}
		printf("verify dif =%f\n", dif);
		sprintf(szPath, "../data/Lena_proc_verify_%dx%d.txt", width, height);
		//imgVerify.SaveText(szPath);
		sprintf(szPath, "../data/Lena_proc_verify(%dx%d)_%dx%d.raw", FILTER_WIDTH, FILTER_HEIGHT, width, height);
		//imgVerify.SaveRaw(szPath);
	}
};

template<typename T>
static int stencil_9pt(int argc, char** argv) {
	DISPLAY_FUNCTION("");
	printf("datatype=double\n");
	int size = 8192; if (argc > 1) size = atoi(argv[1]);
	const int P = 4;
	const int B = 128;
	stencil2d_9pt::Test<T, P, B>(size, size);
	return 0;
}

int stencil_9pt_double(int argc, char** argv) {
	DISPLAY_FUNCTION("");
	printf("datatype=double\n");
	int size = 8192; if (argc > 1) size = atoi(argv[1]);
	const int P = 4;
	const int B = 128;
	stencil2d_9pt::Test<double, P, B>(size, size);
	return 0;
}
/*
int stencil_9pt_float(int argc, char** argv) {
	DISPLAY_FUNCTION("");
	printf("datatype=double\n");
	int size = 8192; if (argc > 1) size = atoi(argv[1]);
	const int P = 4;
	const int B = 128;
	stencil2d_9pt::Test<float, P, B>(size, size);
	return 0;
}
*/


